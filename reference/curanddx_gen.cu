#include "hip/hip_runtime.h"
#include <curanddx.hpp>
#include <iostream>
#include <vector>
#include <chrono>
#include <thread>
#include <cmath>
#include <iomanip>

using RNG = decltype(
    curanddx::Generator<curanddx::philox4_32>() +
    curanddx::PhiloxRounds<10>() +  // Use 10 rounds for better quality
    curanddx::SM<890>() +           // Target RTX 40-series GPU
    curanddx::Thread());            // Thread-level execution

template<class RNG>
__global__ void generate_kernel(
    float4* d_out,
    const unsigned long long seed,
    const typename RNG::offset_type offset,
    const size_t size) {
        
    const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= size / 4) {
        return;
    }

    // Initialize RNG state with seed and offset
    RNG rng(seed, ((offset + i) % 65536), ((offset + i) / 65536));

    // Use uniform distribution [0,1)
    curanddx::uniform<float> dist(0.0f, 1.0f);

    // Generate 4 numbers at once and store to global memory
    d_out[i] = dist.generate4(rng);
}

int main() {
    const size_t NUM_ELEMENTS = 1000000000ULL;  // 1 billion elements
    const size_t BUFFER_SIZE = NUM_ELEMENTS * sizeof(float);
    
    // Allocate device memory
    float* d_out = nullptr;
    hipError_t err = hipMalloc(&d_out, BUFFER_SIZE);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device memory: " << hipGetErrorString(err) << std::endl;
        return 1;
    }

    // Setup kernel launch parameters
    const unsigned int BLOCK_SIZE = 256;
    const unsigned int NUM_BLOCKS = (NUM_ELEMENTS / 4 + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    // Random seed
    const unsigned long long seed = 12345ULL;
    const typename RNG::offset_type offset = 0ULL;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Prepare for multiple runs
    const int NUM_RUNS = 10;
    std::vector<float> timings(NUM_RUNS);
    
    for (int run = 0; run < NUM_RUNS; run++) {
        // Record start event
        hipEventRecord(start);
        
        // Launch kernel
        generate_kernel<RNG><<<NUM_BLOCKS, BLOCK_SIZE>>>((float4*)d_out, seed + run, offset, NUM_ELEMENTS);
        
        // Record stop event
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        
        // Calculate elapsed time
        hipEventElapsedTime(&timings[run], start, stop);
        
        // Check for kernel launch errors
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
            hipFree(d_out);
            return 1;
        }
    }

    // Calculate statistics
    float sum = 0.0f;
    for (float t : timings) {
        sum += t;
    }
    float mean = sum / NUM_RUNS;

    float variance = 0.0f;
    for (float t : timings) {
        variance += (t - mean) * (t - mean);
    }
    variance /= NUM_RUNS;
    float stddev = std::sqrt(variance);

    // Print performance statistics
    std::cout << "Performance Statistics:" << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Average time: " << mean << " ms" << std::endl;
    std::cout << "Standard deviation: " << stddev << " ms" << std::endl;

    // Cleanup timing events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Optional: Verify first few numbers
    std::vector<float> h_verify(1000);
    hipMemcpy(h_verify.data(), d_out, 1000 * sizeof(float), hipMemcpyDeviceToHost);
    
    std::cout << "First 10 random numbers generated:" << std::endl;
    for (int i = 0; i < 10; i++) {
        std::cout << h_verify[i] << " ";
    }
    std::cout << std::endl;

    // Cleanup
    hipFree(d_out);
    return 0;
} 